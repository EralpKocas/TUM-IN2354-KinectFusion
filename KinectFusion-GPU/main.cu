#include "hip/hip_runtime.h"
#include <iostream>
#include <iostream>
#include <fstream>
#include <string>

#include <math.h>
#include "common_functions.h"
#include "data_types.h"
#include "VirtualSensor_freiburg.h"
//#include <opencv2/core/mat.hpp>
//#include "opencv2/imgproc/imgproc.hpp"
//#include <opencv2/highgui.hpp>
#include "surface_measurement.h"

int main() {
    std::cout << "Hello, World!" << std::endl; std::string filenameIn = "/home/ilteber/data/rgbd_dataset_freiburg1_xyz/";

    // load video
    std::cout << "Initialize virtual sensor..." << std::endl;
    bool isFirstFrame = true;
    VirtualSensor_freiburg sensor;
    if (!sensor.init(filenameIn))
    {
        std::cout << "Failed to initialize the sensor!\nCheck file path!" << std::endl;
        return -1;
    }
    ImageConstants img_constants = {
            sensor.getDepthIntrinsics().coeffRef(0, 0),
            sensor.getDepthIntrinsics().coeffRef(1,1),
            sensor.getDepthIntrinsics().coeffRef(0,2),
            sensor.getDepthIntrinsics().coeffRef(1,2),
            sensor.getTrajectory(),
            sensor.getTrajectory().inverse(),
            sensor.getDepthIntrinsics(),
            sensor.getDepthExtrinsics(),
            sensor.getColorImageWidth(),
            sensor.getColorImageHeight(),
            sensor.getDepthImageWidth(),
            sensor.getDepthImageHeight(),
    };
    while (sensor.processNextFrame()) {
        ImageData img_data = {
                sensor.getDepthImageWidth(),
                sensor.getDepthImageHeight(),
                cv::cuda::GpuMat(sensor.getDepthImageWidth(), sensor.getDepthImageHeight(), CV_32F, sensor.getDepth()),
                cv::cuda::GpuMat(sensor.getColorImageWidth(), sensor.getColorImageHeight(), CV_8U, sensor.getColorRGBX()),
                //cv::cuda::GpuMat(640, 480, CV_32F, sensor.getDepth()),
                //cv::cuda::GpuMat(640, 480, CV_8U, sensor.getColorRGBX()),
        };
        // TODO: inverse trajectory is nan for all indices. check!

        SurfaceLevelData surf_data = {
                3,
                img_constants.m_colorImageWidth,
                img_constants.m_colorImageHeight,
                img_constants.fX,
                img_constants.fY,
                img_constants.cX,
                img_constants.cY,
        };

//        cv::Mat result;
//        img_data.m_depthMap.download(result);
//        cv::imshow("result", result);
//        cv::waitKey(30);

        //std::cout << "line 50: "  << result << std::endl;
        //std::cout << "line 51: "  << img_data.m_colorMap << std::endl;

        // step 1: Surface Measurement
        surface_measurement_pipeline(&surf_data, img_data);

        // step 2: Pose Estimation, for frame == 0, don't perform
        if(!isFirstFrame){

        }else{
            isFirstFrame = false;
        }
        // step 3: Surface Reconstruction Update
        // step 4: Raycast Prediction
    }
    return 0;
}
