#include "hip/hip_runtime.h"
//
// Created by ilteber on 05.07.21.
//
#include "data_types.h"

#include "common_functions.h"
#include "surface_reconstruction.h"

////HELPER FUNCTIONS
__device__ float calculateLambda( Matrix4f depth_ext,
                                  Matrix4f m_traj,
                                  Matrix3f depth_intrinsics, Matrix3f intrinsicsInv, Vector3f p){
    Vector2i projected = perspective_projection(depth_ext, m_traj, depth_intrinsics, p);
    Vector3f dot_p = Vector3f(projected.x(), projected.y(), 1.0f);
    return (intrinsicsInv * dot_p).norm();
}
//

__device__ Vector2i perspective_projection(Matrix4f depth_ext,
                                           Matrix4f m_traj,
                                           Matrix3f depth_intrinsics, Vector3f p)
{
    Vector4f p_temp = Vector4f(p.x(), p.y(), p.z(), 1.0);
    Matrix4f identity = Matrix4f::Zero();
    identity.block<3, 3>(0, 0) = Matrix3f::Identity();
    Vector3f p2 = depth_intrinsics * identity.block<3, 4>(0, 0) * depth_ext * m_traj * p_temp;
    return Vector2i((int) round(p2.x() / p2.z()), (int) round(p2.y() / p2.z()));
}
//
//
__device__ float calculateSDF_truncation(float truncation_distance, float sdf){
    if (sdf >= -truncation_distance) {
        return fmin(1.f, sdf / truncation_distance) * (sdf < 0.f ? -1.f : sdf > 0.f); // determine threshold, 1.f currently
    }
    else return -1.f; // return - of threshold
}
//
////λ = ||K^-1*x||2
__device__ float calculateCurrentTSDF(Matrix4f pose_traj, Matrix4f depth_ext,
                                      Matrix4f m_traj, Matrix3f depth_intrinsics, float depth,
                                      Matrix3f intrinsics, Vector3f p, float truncation_distance){
    float current_tsdf = -1.f * ((1.f / calculateLambda(depth_ext, m_traj, depth_intrinsics, intrinsics, p)) *
            (pose_traj.block<3, 1>(0, 3) - p).norm() - depth);
    return calculateSDF_truncation(truncation_distance, current_tsdf);
}
//
//
//// calculate weighted running tsdf average
__device__ float calculateWeightedTSDF(int current_weight, float current_tsdf, int new_weight, float new_tsdf){
    float updated_tsdf = (current_weight * current_tsdf + new_weight * new_tsdf) /
                         (current_weight + new_weight);
    return updated_tsdf;
}
//
//// calculate weighted running weight average
__device__ int calculateWeightedAvgWeight(int current_weight, int new_weight){
    return current_weight + new_weight;
}
//
//// truncate updated weight
__device__ int calculateTruncatedWeight(int weighted_avg, int some_value){
    if(weighted_avg < some_value)
        return weighted_avg;
    return some_value;
//    return std::min(weighted_avg, some_value);
}
//
__device__ Vector4uc calculateWeightedColorUpdate(int current_weight, Vector4uc curr_color,
                                                  int new_weight, Vector4uc new_color)
{
    return Vector4uc((current_weight * curr_color[0] + new_weight * new_color[0]) /
                     (current_weight + new_weight),
                     (current_weight * curr_color[1] + new_weight * new_color[1]) /
                     (current_weight + new_weight),
                     (current_weight * curr_color[2] + new_weight * new_color[2]) /
                     (current_weight + new_weight),
                     (current_weight * curr_color[3] + new_weight * new_color[3]) /
                     (current_weight + new_weight));
}

__global__ void updateSurfaceReconstructionGlobal(ImageConstants*& imageConstants,
                                 cv::cuda::PtrStepSz<float> tsdf_values,
                                 cv::cuda::PtrStepSz<float> tsdf_weight,
                                 cv::cuda::PtrStepSz<Vector4uc> tsdf_color,
                                 cv::cuda::PtrStepSz<Vector4uc> color_map,
                                 cv::cuda::PtrStepSz<float> depth_map,
                                 int width, int height,
                                 float voxel_scale,
                                 int volume_size,
                                 Matrix4f depth_ext,
                                 Matrix4f m_traj,
                                 Matrix3f depth_intrinsics,
                                 Matrix3f depth_intrinsics_inv,
                                 Matrix4f pose_traj,
                                 float truncation_distance){

    int threadX = threadIdx.x + blockDim.x * blockIdx.x;
    if (threadX >= width or threadX < 0)
        return;

    int threadY = threadIdx.y + blockDim.y * blockIdx.y;
    if (threadY >= height or threadY < 0)
        return;

    int truncate_updated_weight = 128;

    for(int k=0; k < volume_size; ++k) {
        const Vector3f global_coord((static_cast<float>(threadX) + 0.5f) * voxel_scale,
                               (static_cast<float>(threadY) + 0.5f) * voxel_scale,
                               (static_cast<float>(k) + 0.5f) * voxel_scale);

        Vector3f camera_coord = (depth_ext * m_traj *
                                 Vector4f(global_coord.x(),
                                          global_coord.y(), global_coord.z(), 1.0f)).block<3, 1>(0, 0);

        if (camera_coord.z() <= 0) continue;

        Vector2i image_coord = perspective_projection(depth_ext, m_traj, depth_intrinsics, global_coord);

        if (image_coord.x() < 0 || image_coord.x() >= width
            || image_coord.y() < 0 || image_coord.y() >= height)
            continue;

        float depth = depth_map.ptr((int) image_coord.y())[image_coord.x()];

        if (depth <= 0) continue;

        float F_rk = calculateCurrentTSDF(pose_traj, depth_ext, m_traj, depth_intrinsics, depth,
                                          depth_intrinsics_inv, global_coord, truncation_distance);

        printf("3333\n");

        if (F_rk == -1.f) continue;
        printf("4444\n");

        int W_k = 1;
        // TODO: it should be y, if y!=z, change it volume_y
        int prev_weight = tsdf_weight.ptr(k * volume_size + threadY)[threadX];
        int prev_tsdf = tsdf_values.ptr(k * volume_size + threadY)[threadX];

        float updated_tsdf = calculateWeightedTSDF(prev_weight, prev_tsdf, W_k, F_rk);

        printf("5555\n");

        int updated_W_k = calculateWeightedAvgWeight(prev_weight, W_k);
        int truncated_weight = calculateTruncatedWeight(updated_W_k, truncate_updated_weight);

        // TODO: it should be y, if y!=z, change it volume_y
        tsdf_values.ptr(k * volume_size + threadY)[threadX] = updated_tsdf;
        tsdf_weight.ptr(k * volume_size + threadY)[threadX] = truncated_weight;

        printf("7777\n");

        Vector4uc curr_color;
        if (F_rk <= truncation_distance / 2 &&
            F_rk >= -truncation_distance / 2) {
            // TODO: it should be y, if y!=z, change it volume_y
            Vector4uc prev_color = tsdf_color.ptr(k * volume_size + threadY)[threadX];
            Vector4uc image_color = color_map.ptr(image_coord.y())[image_coord.x()];
            curr_color = calculateWeightedColorUpdate(prev_weight, prev_color,
                                                      truncated_weight, image_color);
            // TODO: it should be y, if y!=z, change it volume_y
            tsdf_color.ptr(k * volume_size + threadY)[threadX] = curr_color;

        }

      }
}
void updateSurfaceReconstruction(Pose* pose,ImageConstants* imageConstants,
                                                  ImageData* imageData, SurfaceLevelData* surf_data,  GlobalVolume* global_volume)
{
//    printf("1234\n");

    const dim3 threads(8, 8);
    const dim3 blocks((global_volume->volume_size.x + threads.x - 1) / threads.x,
                      (global_volume->volume_size.y + threads.y - 1) / threads.y);
    cv::cuda::GpuMat& tsdf_vals = global_volume->TSDF_values;
    cv::cuda::GpuMat& tsdf_weights = global_volume->TSDF_weight;
    cv::cuda::GpuMat& tsdf_color = global_volume->TSDF_color;
    cv::cuda::GpuMat& color_map = imageData->m_colorMap;
    cv::cuda::GpuMat& depth_map = imageData->m_depthMap;
//    printf("5678\n");

    updateSurfaceReconstructionGlobal<<<blocks,threads>>>(imageConstants,
                                     tsdf_vals,tsdf_weights, tsdf_color,
                                     color_map, depth_map,
                                     imageConstants->m_colorImageWidth,
                                     imageConstants->m_colorImageHeight,
                                     global_volume->voxel_scale,
                                     global_volume->volume_size.z,
                                     imageConstants->m_depthExtrinsics,
                                     imageConstants->m_trajectory,
                                     imageConstants->m_depthIntrinsics,
                                     imageConstants->m_depthIntrinsicsInv,
                                     pose->m_trajectory,
                                     global_volume->truncation_distance);
    // debugging purposes.
    printf("7890\n");
    cv::Mat result;
    tsdf_vals.download(result);
    std::cout << result.at<float>(0, 0) << std::endl;
    assert(hipSuccess == hipDeviceSynchronize());

}

