#include "hip/hip_runtime.h"
#include "data_types.h"
#include "surface_prediction.h"

//WARNING: Volume stuff might not have converted to CUDA right.

__device__ Vector3f calculate_pixel_ray_cast(Matrix3f rotation, Vector3f translation, ImageConstants imageConstants) {
    int threadX = blockIdx.x * blockDim.x + threadIdx.x;
    int threadY = blockIdx.y * blockDim.y + threadIdx.y;

    //img to cam
    float camera_x = (threadX - imageConstants.cX) / imageConstants.fX;
    float camera_y = (threadY - imageConstants.cY) / imageConstants.fY;
    Vector3f camera_vec = Vector3f(camera_x, camera_y, 1.f);
    //cam to global
    Vector3f ray_cast = rotation * camera_vec + translation;

    return ray_cast;
}

// P = O + t * R where R is normalized ray direction and O is eye, translation vector in our case.
// Then, t = (P - O) / R
__device__ float calculate_search_length(Vector3f eye, Vector3f ray_dir) {
    int threadX = blockIdx.x * blockDim.x + threadIdx.x;
    int threadY = blockIdx.y * blockDim.y + threadIdx.y;
    int threadZ = blockIdx.z * blockDim.z + threadIdx.z;
    float t_x = (threadX - eye.x()) / ray_dir.x();
    float t_y = (threadY - eye.y()) / ray_dir.y();
    float t_z = (threadZ - eye.z()) / ray_dir.z();

    return fmax(fmax(fabs(t_x), fabs(t_y)), fabs(t_z));
}

__device__ bool gridInVolume(GlobalVolume* global_volume, Vector3f curr_grid) {
    int dx = global_volume->getDimX();
    int dy = global_volume->getDimY();
    int dz = global_volume->getDimZ();
    return !(curr_grid.x() < 1 || curr_grid.x() >= dx - 1 ||
             curr_grid.y() < 1 || curr_grid.y() >= dy - 1 ||
             curr_grid.z() < 1 || curr_grid.z() >= dz - 1);
}

__device__ float calculate_trilinear_interpolation(GlobalVolume* global_volume, Vector3f p) {
    Vector3i p_int = Vector3i((int) p.x(), (int) p.y(), (int) p.z());

    //couldn't find a way to do this one
    float c000 = global_volume->get(p_int.x(), p_int.y(), p_int.z()).tsdf_distance_value;
    float c001 = global_volume->get(p_int.x(), p_int.y(), p_int.z() + 1).tsdf_distance_value;
    float c010 = global_volume->get(p_int.x(), p_int.y() + 1, p_int.z()).tsdf_distance_value;
    float c011 = global_volume->get(p_int.x(), p_int.y() + 1, p_int.z() + 1).tsdf_distance_value;
    float c100 = global_volume->get(p_int.x() + 1, p_int.y(), p_int.z()).tsdf_distance_value;
    float c101 = global_volume->get(p_int.x() + 1, p_int.y(), p_int.z() + 1).tsdf_distance_value;
    float c110 = global_volume->get(p_int.x() + 1, p_int.y() + 1, p_int.z()).tsdf_distance_value;
    float c111 = global_volume->get(p_int.x() + 1, p_int.y() + 1, p_int.z() + 1).tsdf_distance_value;

    float xd = p.x() - p_int.x();
    float yd = p.y() - p_int.y();
    float zd = p.z() - p_int.z();

    float c00 = c000 * (1 - xd) + c100 * xd;
    float c01 = c001 * (1 - xd) + c101 * xd;
    float c10 = c010 * (1 - xd) + c110 * xd;
    float c11 = c011 * (1 - xd) + c111 * xd;

    float c0 = c00 * (1 - yd) + c10 * yd;
    float c1 = c01 * (1 - yd) + c11 * yd;

    float c = c0 * (1 - zd) + c1 * zd;

    return c;
}

__global__ void helper_compute_normal_map(int width, int height) {
    int threadX = threadIdx.x + blockDim.x * blockIdx.x;
    if (threadX >= width or threadX < 0)
        return;
    int threadY = threadIdx.y + blockDim.y * blockIdx.y;
    if (threadY >= height or threadY < 0)
        return;

    Vector3f curr_vertex = vertex_map_predicted.ptr(threadY)[threadX];

    if (curr_vertex.z() == 0.f || isnan(curr_vertex.z())) {
        normal_map_predicted.ptr(threadY)[threadX] = Vector3f(0.f, 0.f, 0.f);
    } else {
        Vector3f neigh_1 = Vector3f(vertex_map_predicted.ptr(threadY - 1)[threadX].x() -
                                    vertex_map_predicted.ptr(threadY + 1)[threadX].x(),
                                    vertex_map_predicted.ptr(threadY - 1)[threadX].y() -
                                    vertex_map_predicted.ptr(threadY + 1)[threadX].y(),
                                    vertex_map_predicted.ptr(threadY - 1)[threadX].z() -
                                    vertex_map_predicted.ptr(threadY + 1)[threadX].z());

        Vector3f neigh_2 = Vector3f(vertex_map_predicted.ptr(threadY)[threadX - 1].x() -
                                    vertex_map_predicted.ptr(threadY)[threadX + 1].x(),
                                    vertex_map_predicted.ptr(threadY)[threadX - 1].y() -
                                    vertex_map_predicted.ptr(threadY)[threadX + 1].y(),
                                    vertex_map_predicted.ptr(threadY)[threadX - 1].z() -
                                    vertex_map_predicted.ptr(threadY)[threadX + 1].z());

        Vector3f cross_prod = neigh_1.cross(neigh_2);
        cross_prod.normalize();
        if (cross_prod.z() > 0) cross_prod *= -1;
        normal_map_predicted.ptr(threadY)[threadX] = cross_prod;
    }
}

__global__ void predict_surface(GlobalVolume global_volume, Pose pose,
                                cv::cuda::PtrStep<Vector3f> vertex_map,
                                cv::cuda::PtrStep<Vector3f> normal_map,
                                cv::cuda::PtrStep<Vector4uc> color_map,
                                float fX, float fY, float cX, float cY,
                                int width, int height, int level) {

    //simple ray skipping (speedup):
    //near F(p)=0, the fused volume holds a good approx to true sdf from p to the nearest surf interface.
    //so using known trunc dist, march along the ray in staps size < mu while F(p) vals have +ve trunc vals

    //higher quality intersections by ray/trilin cell intersection (simple approx):
    //find param t* at which the intersect more precise: t*=t-(deltat Ft+)/(F(t+deltat)+ - Ft+)

    //predicted vertex and normal maps are computed at the interpolated location in the global frame.

    int threadX = threadIdx.x + blockDim.x * blockIdx.x;
    if (threadX >= width or threadX < 0)
        return;

    int threadY = threadIdx.y + blockDim.y * blockIdx.y;
    if (threadY >= height or threadY < 0)
        return;

    float step_size = global_volume.truncation_distance;

    //per pixel raycast. march start from the min depth, stop with zero crossing or back face
    // +0.5 for reaching pixel centers
    Vector3f pixel_ray = calculate_pixel_raycast(rotation, translation, image_constants);
    float camera_x = ((float) (threadX + 0.5) - surf_data.level_cX) / surf_data.level_fX;  // image to camera
    float camera_y = ((float) (threadY + 0.5) - surf_data.level_cY) / surf_data.level_fY;  // image to camera
    Vector3f pixel = Vector3f(camera_x, camera_y, 1.f);

    //for point on or close to surf. interface Fk(p)=0, gradient(Fk(p))=orthogonal to zero level set.
    //so the surface normal of pixel u along which p was found deriv of SDF: v[grad(F(p))].
    //Question: Why do we calculate two of them?
    Vector3f ray_dir_2 = (rotation * pixel).normalized();
    Vector3f ray_dir = (pixel_ray - translation).normalized();

    //then scale the deriv in each dimension.
    //min and max rendering range [0.4,8] => bounded time per pixel computation for any size or complexity
    //of scene with a fixed vol resolution
    //abimiz burda get_min get_max time fonksiyonlari kullanmis bizde yok bu
    //float t = calculate_search_length(translation, pixel_ray, ray_dir);  // t
    float max_ray_length = Vector3i(global_volume->getDimX(),
                                    global_volume->getDimY(),
                                    global_volume->getDimZ()).norm();

    //Vector3f pixel_grid = global_volume->compute_grid(pixel_ray);
    //Vector3f ray_dir_grid = global_volume->compute_grid(ray_dir);

    Vector3f init_pos = Vector3f(0.f, 0.f, 0.f);
    for (float step = 0; step < max_ray_length; step += step_size * 0.5) {
        Vector3f curr_pos = translation + (float) step * ray_dir;
        Vector3f curr_grid = global_volume->compute_grid(curr_pos);

        if (!gridInVolume(global_volume, curr_grid)) continue;
        init_pos = curr_pos;
        break;
    }

    if(!init_pos.allFinite() || init_pos.x() == 0.f ||
    init_pos.y() == 0.f || init_pos.z() == 0.f) continue;

    Vector3f eye_grid = global_volume->compute_grid(init_pos);
    float tsdf = global_volume->
            get((int) eye_grid.x(), (int) eye_grid.y(), (int) eye_grid.z()).tsdf_distance_value;

    float prev_tsdf = tsdf;
    Vector3f prev_grid = eye_grid; // TODO: check this, something is seem bad!

    for (float step = 0; step < max_ray_length; step += step_size * 0.5) {
        //Vector3f curr_grid = eye_grid + (float) step * ray_dir_grid;
        Vector3f curr_pos = init_pos + (float) step * ray_dir;
        Vector3f curr_grid = global_volume->compute_grid(curr_pos);

        if (!gridInVolume(image_properties, global_volume, curr_grid)) continue;

        float curr_tsdf = global_volume->
                get((int) curr_grid.x(), (int) curr_grid.y(), (int) curr_grid.z()).tsdf_distance_value;

        if (prev_tsdf < 0.f && curr_tsdf > 0.f) break;  // zero-crossing from behind

        if (prev_tsdf > 0.f && curr_tsdf < 0.f)  // zero-crossing is found
            {
            float prev_tri_interpolated_sdf = calculate_trilinear_interpolation(global_volume,
                                                                                prev_grid);
            float curr_tri_interpolated_sdf = calculate_trilinear_interpolation(global_volume,
                                                                                curr_grid);

            Voxel before = global_volume->get(curr_grid.x(), curr_grid.y(), curr_grid.z());
            global_volume->set(translation.x(), translation.y(), translation.z(),
                               global_volume->set_occupied(curr_grid));
            Voxel after = global_volume->get(curr_grid.x(), curr_grid.y(), curr_grid.z());

            //float t_star = step - ((step_size * 0.5f * prev_tsdf)/ (curr_tsdf - prev_tsdf));
            // t_star = t - ((step_size * prev_tsdf) / (curr_tsdf - prev_tsdf))

            float t_star = step - ((step_size * 0.5f * prev_tri_interpolated_sdf)
                    / (curr_tri_interpolated_sdf - prev_tri_interpolated_sdf));

            Vector3f grid_location = translation + t_star * ray_dir;

            if (!gridInVolume(global_volume, grid_location)) break;

            Vector3f vertex = translation + t_star * ray_dir;

            vertex_map_predicted.ptr(threadY)[threadX] = vertex;
            }
        prev_tsdf = curr_tsdf;
        prev_grid = curr_grid;
    }
    helper_compute_normal_map(width, height);

}

void surface_prediction(SurfaceLevelData* surf_data, GlobalVolume global_volume, Pose pose){
    //for (int i = 0; i < surf_data->level; i++) {
        //did not change the name convention. Commented out the for loop.
        //changed this from (8,8)
        dim3 block(32, 32);

        float cols = surf_data->level_img_width[i];
        float rows = surf_data->level_img_height[i];

        //Define zero level set for models Fk=tsdf val=0
        //Question: we never set these to 0?
        cv::cuda::GpuMat& vertex_map = surf_data->vertex_map_predicted[i];
        cv::cuda::GpuMat& normal_map = surf_data->normal_map_predicted[i];
        cv::cuda::GpuMat& color_map = surf_data->color_map[i];

        float fX = surf_data->level_fX[i];
        float fY = surf_data->level_fY[i];
        float cX = surf_data->level_cX[i];
        float cY = surf_data->level_cY[i];

        dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
        predict_surface<<<grid, block>>>(global_volume, pose,
                                         vertex_map,
                                         normal_map,
                                         color_map,
                                         fX, fY, cX, cY,
                                         cols, rows, i);

        hipDeviceSynchronize();
    //}
}